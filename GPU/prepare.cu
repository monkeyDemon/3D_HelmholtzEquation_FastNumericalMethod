#include "hip/hip_runtime.h"
#include "prepare.h"
#include "stdio.h"
#include "math.h"
#define PI 3.14159265358979323846

//void computeBU_Z_array(float *BU,int i,int j,int M,int N,int K,float h,float K0);

__host__ __device__ float compute_BoundaryCondition(int M,float h,int i,int j,int k)
{
	float Xstart=0;
	float Ystart=0;
	float Zstart=0;
	
	float x=Xstart+h*i;
	float y=Ystart+h*j;
	float z=Zstart+h*k;
       
	float value=0;	
	
	if(i==0)
	{
	        value=sin(PI*y)*sin(PI*z);
		//printf("x=0,y=%f,z=%f,value:%f\n",y,z,value);
	}
	else if(i==M+1)
	{
	        value=2*sin(PI*y)*sin(PI*z);
		//printf("x=1,y=%f,z=%f,value:%f\n",y,z,value);
	}
	else
	        value=0;
	
	return value;
}


__global__ void computeBU_kernel(float* BU, int size, float h, float K0)
{
	int i=blockIdx.x;
	int j=threadIdx.x;
	
	//printf("%d  %d\n",i,j);

	if(i>=size||j>=size)
		return;
	
	i++;
	j++;	
	
	//if(i==1)
	//	printf("i=%d\n",i);
	//if(i==128)
	//	printf("i=%d\n",i);
	//if(i==129)
	//	printf("!!!i=%d\n",i);

	//computeBU_Z_array(BU,i,j,size,size,size,h,K0);

	int M=size;
	int N=size;
	int K=size;

	int Idx=(i-1)*N*K+ (j-1)*K -1;

	float para1=(1+h*h*K0*K0/12)/(h*h);
	float para2=1/(6*h*h);

	float temp=0;	
	float temp_gama=0;
	
	if( i==1 && j==1  )
	{
		for(int k=1;k<=K;k++)
		{
			if(k==1)
			{
				temp=compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,i,0,k)+compute_BoundaryCondition(M,h,i,j,0);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,0,0,k)+compute_BoundaryCondition(M,h,0,j,0)-4*compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,0,j,k+1)+compute_BoundaryCondition(M,h,0,j+1,k)+compute_BoundaryCondition(M,h,i,0,0)-4*compute_BoundaryCondition(M,h,i,0,k)+compute_BoundaryCondition(M,h,i,0,k+1)-4*compute_BoundaryCondition(M,h,i,j,0)+compute_BoundaryCondition(M,h,i,j+1,0)+compute_BoundaryCondition(M,h,i+1,0,k)+compute_BoundaryCondition(M,h,i+1,j,0);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
			else if(k==K)
			{
				temp=compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,i,0,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j,K+1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
				temp=compute_BoundaryCondition(M,h,0,0,k)+compute_BoundaryCondition(M,h,0,j,k-1)-4*compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,0,j,k+1)+compute_BoundaryCondition(M,h,0,j+1,k)+compute_BoundaryCondition(M,h,i,0,k-1)-4*compute_BoundaryCondition(M,h,i,0,k)+compute_BoundaryCondition(M,h,i,0,k+1)+compute_BoundaryCondition(M,h,i+1,0,k);
				temp_gama=-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
			}
			else
			{
				temp=compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,i,0,k);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,0,0,k)+compute_BoundaryCondition(M,h,0,j,k-1)-4*compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,0,j,k+1)+compute_BoundaryCondition(M,h,0,j+1,k)+compute_BoundaryCondition(M,h,i,0,k-1)-4*compute_BoundaryCondition(M,h,i,0,k)+compute_BoundaryCondition(M,h,i,0,k+1)+compute_BoundaryCondition(M,h,i+1,0,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
		}
	}
	else if( i==1 && j==N  )
	{
		for(int k=1;k<=K;k++)
		{
			if(k==1)
			{
				temp=compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k-1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
			else if(k==K)
			{
				temp=compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
			}
			else
			{
				temp=compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
		}
	}
	else if( i==1 && j>1 && j<N  )
	{
		for(int k=1;k<=K;k++)
		{
			if(k==1)
			{
				temp=compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
			else if(k==K)
			{
				temp=compute_BoundaryCondition(M,h,i-1,j,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
			}
			else
			{
				temp=compute_BoundaryCondition(M,h,i-1,j,k);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
		}
	}
	else if( i==M && j==1 )
	{
		for(int k=1;k<=K;k++)
		{
			if(k==1)
			{
				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
			else if(k==K)
			{
				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j-1,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
			}
			else
			{
				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j-1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
		}
	}
	else if( i==M && j==N )
	{
		for(int k=1;k<=K;k++)
		{
			if(k==1)
			{
				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
			else if(k==K)
			{
				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
			}
			else
			{
				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
		}
	}
	else if( i==M && j>1 && j<N  )
	{
		for(int k=1;k<=K;k++)
		{
			if(k==1)
			{
				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
			else if(k==K)
			{
				temp=compute_BoundaryCondition(M,h,i+1,j,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
				temp=compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
			}
			else
			{
				temp=compute_BoundaryCondition(M,h,i+1,j,k);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
		}
	}
	else if( i>1 && i<M && j==1  )
	{
		for(int k=1;k<=K;k++)
		{	
			if(k==1)
			{
				temp=compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
			else if(k==K)
			{
				temp=compute_BoundaryCondition(M,h,i,j-1,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k);
				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
			}
			else
			{
				temp=compute_BoundaryCondition(M,h,i,j-1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
		}
	}
	else if( i>1 && i<M && j==N   )
	{
		for(int k=1;k<=K;k++)
		{
			if(k==1)
			{
				temp=compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k-1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
			else if(k==K)
			{
				temp=compute_BoundaryCondition(M,h,i,j+1,k);
				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
			}
			else
			{
				temp=compute_BoundaryCondition(M,h,i,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
		}
	}
	else if( i>1 && i<M && j>1 && j<N )
	{
		for(int k=1;k<=K;k++)
		{
			if(k==1)
			{
				temp=compute_BoundaryCondition(M,h,i,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para1;
				temp=compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j,k-1);
				BU[Idx+k]=BU[Idx+k]+temp*para2;
			}
			else if(k==K)
			{
				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
			}
			else
			{
				// do nothing
			}
		}
	}
}


void computeBU(float* BU, int size, float h, float K0)
{
	float *dev_BU=0;

	hipMalloc(&dev_BU,sizeof(float)*size*size*size);

	hipMemset(dev_BU,0,sizeof(float)*size*size*size);

	computeBU_kernel<<<size,size>>>(dev_BU,size,h,K0);

	hipMemcpy(BU,dev_BU,sizeof(float)*size*size*size,hipMemcpyDeviceToHost);

	hipFree(dev_BU);
}


//__host__ __device__ void computeBU_Z_array(float *BU,int i,int j,int M,int N,int K,float h,float K0)
//{
//	int Idx=(i-1)*N*K+ (j-1)*K -1;
//
//	float para1=(1+h*h*K0*K0/12)/(h*h);
//	float para2=1/(6*h*h);
//
//	float temp=0;	
//	float temp_gama=0;
//	
//	if( i==1 && j==1  )
//	{
//		for(int k=1;k<=K;k++)
//		{
//			if(k==1)
//			{
//				temp=compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,i,0,k)+compute_BoundaryCondition(M,h,i,j,0);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,0,0,k)+compute_BoundaryCondition(M,h,0,j,0)-4*compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,0,j,k+1)+compute_BoundaryCondition(M,h,0,j+1,k)+compute_BoundaryCondition(M,h,i,0,0)-4*compute_BoundaryCondition(M,h,i,0,k)+compute_BoundaryCondition(M,h,i,0,k+1)-4*compute_BoundaryCondition(M,h,i,j,0)+compute_BoundaryCondition(M,h,i,j+1,0)+compute_BoundaryCondition(M,h,i+1,0,k)+compute_BoundaryCondition(M,h,i+1,j,0);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//			else if(k==K)
//			{
//				temp=compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,i,0,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j,K+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
//				temp=compute_BoundaryCondition(M,h,0,0,k)+compute_BoundaryCondition(M,h,0,j,k-1)-4*compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,0,j,k+1)+compute_BoundaryCondition(M,h,0,j+1,k)+compute_BoundaryCondition(M,h,i,0,k-1)-4*compute_BoundaryCondition(M,h,i,0,k)+compute_BoundaryCondition(M,h,i,0,k+1)+compute_BoundaryCondition(M,h,i+1,0,k);
//				temp_gama=-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
//			}
//			else
//			{
//				temp=compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,i,0,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,0,0,k)+compute_BoundaryCondition(M,h,0,j,k-1)-4*compute_BoundaryCondition(M,h,0,j,k)+compute_BoundaryCondition(M,h,0,j,k+1)+compute_BoundaryCondition(M,h,0,j+1,k)+compute_BoundaryCondition(M,h,i,0,k-1)-4*compute_BoundaryCondition(M,h,i,0,k)+compute_BoundaryCondition(M,h,i,0,k+1)+compute_BoundaryCondition(M,h,i+1,0,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//		}
//	}
//	else if( i==1 && j==N  )
//	{
//		for(int k=1;k<=K;k++)
//		{
//			if(k==1)
//			{
//				temp=compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k-1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//			else if(k==K)
//			{
//				temp=compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
//			}
//			else
//			{
//				temp=compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//		}
//	}
//	else if( i==1 && j>1 && j<N  )
//	{
//		for(int k=1;k<=K;k++)
//		{
//			if(k==1)
//			{
//				temp=compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//			else if(k==K)
//			{
//				temp=compute_BoundaryCondition(M,h,i-1,j,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
//			}
//			else
//			{
//				temp=compute_BoundaryCondition(M,h,i-1,j,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)-4*compute_BoundaryCondition(M,h,i-1,j,k)+compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i-1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//		}
//	}
//	else if( i==M && j==1 )
//	{
//		for(int k=1;k<=K;k++)
//		{
//			if(k==1)
//			{
//				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//			else if(k==K)
//			{
//				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j-1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
//			}
//			else
//			{
//				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j-1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//		}
//	}
//	else if( i==M && j==N )
//	{
//		for(int k=1;k<=K;k++)
//		{
//			if(k==1)
//			{
//				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//			else if(k==K)
//			{
//				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
//			}
//			else
//			{
//				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//		}
//	}
//	else if( i==M && j>1 && j<N  )
//	{
//		for(int k=1;k<=K;k++)
//		{
//			if(k==1)
//			{
//				temp=compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//			else if(k==K)
//			{
//				temp=compute_BoundaryCondition(M,h,i+1,j,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
//				temp=compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
//			}
//			else
//			{
//				temp=compute_BoundaryCondition(M,h,i+1,j,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1)-4*compute_BoundaryCondition(M,h,i+1,j,k)+compute_BoundaryCondition(M,h,i+1,j,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//		}
//	}
//	else if( i>1 && i<M && j==1  )
//	{
//		for(int k=1;k<=K;k++)
//		{	
//			if(k==1)
//			{
//				temp=compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j-1,k)+compute_BoundaryCondition(M,h,i+1,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//			else if(k==K)
//			{
//				temp=compute_BoundaryCondition(M,h,i,j-1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
//			}
//			else
//			{
//				temp=compute_BoundaryCondition(M,h,i,j-1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j-1,k)+compute_BoundaryCondition(M,h,i,j-1,k+1)+compute_BoundaryCondition(M,h,i+1,j-1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//		}
//	}
//	else if( i>1 && i<M && j==N   )
//	{
//		for(int k=1;k<=K;k++)
//		{
//			if(k==1)
//			{
//				temp=compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k-1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//			else if(k==K)
//			{
//				temp=compute_BoundaryCondition(M,h,i,j+1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
//			}
//			else
//			{
//				temp=compute_BoundaryCondition(M,h,i,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k-1)-4*compute_BoundaryCondition(M,h,i,j+1,k)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j+1,k);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//		}
//	}
//	else if( i>1 && i<M && j>1 && j<N )
//	{
//		for(int k=1;k<=K;k++)
//		{
//			if(k==1)
//			{
//				temp=compute_BoundaryCondition(M,h,i,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para1;
//				temp=compute_BoundaryCondition(M,h,i-1,j,k-1)+compute_BoundaryCondition(M,h,i,j-1,k-1)-4*compute_BoundaryCondition(M,h,i,j,k-1)+compute_BoundaryCondition(M,h,i,j+1,k-1)+compute_BoundaryCondition(M,h,i+1,j,k-1);
//				BU[Idx+k]=BU[Idx+k]+temp*para2;
//			}
//			else if(k==K)
//			{
//				temp_gama=compute_BoundaryCondition(M,h,i,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para1;
//				temp_gama=compute_BoundaryCondition(M,h,i-1,j,k+1)+compute_BoundaryCondition(M,h,i,j-1,k+1)-4*compute_BoundaryCondition(M,h,i,j,k+1)+compute_BoundaryCondition(M,h,i,j+1,k+1)+compute_BoundaryCondition(M,h,i+1,j,k+1);
//				BU[Idx+k]=BU[Idx+k]+temp_gama*para2;
//			}
//			else
//			{
//				// do nothing
//			}
//		}
//	}
//}
